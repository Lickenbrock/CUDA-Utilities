#include "cublasWrappers.cuh"

/**************/
/* CUBLASTDOT */
/**************/
hipblasStatus_t cublasTdot(hipblasHandle_t handle, int n, const float *x, int incx, const float *y, int incy, float *result) {
	return hipblasSdot(handle, n, x, incx, y, incy, result);
}

hipblasStatus_t cublasTdot(hipblasHandle_t handle, int n, const double *x, int incx, const double *y, int incy, double *result) {
	return hipblasDdot(handle, n, x, incx, y, incy, result);
}

/***************/
/* CUBLASTNRM2 */
/***************/
hipblasStatus_t cublasTnrm2(hipblasHandle_t handle, int n, const float *x, int incx, float *result) {
	return hipblasSnrm2(handle, n, x, incx, result);
}

hipblasStatus_t cublasTnrm2(hipblasHandle_t handle, int n, const double *x, int incx, double *result) {
	return hipblasDnrm2(handle, n, x, incx, result);
}

/***************/
/* CUBLASTSCAL */
/***************/
hipblasStatus_t cublasTscal(hipblasHandle_t handle, int n, const float *alpha, float *x, int incx) {
	return hipblasSscal(handle, n, alpha, x, incx);
}
	
hipblasStatus_t cublasTscal(hipblasHandle_t handle, int n, const double *alpha, double *x, int incx) {
	return hipblasDscal(handle, n, alpha, x, incx);
}
	
/***************/
/* CUBLASTAXPY */
/***************/
hipblasStatus_t cublasTaxpy(hipblasHandle_t handle, int n, const float *alpha, const float *x, int incx, float *y, int incy) {
	return hipblasSaxpy(handle, n, alpha, x, incx, y, incy);
}

hipblasStatus_t cublasTaxpy(hipblasHandle_t handle, int n, const double *alpha, const double *x, int incx, double *y, int incy) {
	return hipblasDaxpy(handle, n, alpha, x, incx, y, incy);
}

/***************/
/* CUBLASTCOPY */
/***************/
hipblasStatus_t cublasTcopy(hipblasHandle_t handle, int n, const float *x, int incx, float *y, int incy) {
	return hipblasScopy(handle, n, x, incx, y, incy);
}

hipblasStatus_t cublasTcopy(hipblasHandle_t handle, int n, const double *x, int incx, double *y, int incy) {
	return hipblasDcopy(handle, n, x, incx, y, incy);
}

 
